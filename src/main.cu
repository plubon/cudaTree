////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/* Template project which demonstrates the basics on how to setup a project
* example application.
* Host code.
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <iostream>
// includes CUDA
#include <hip/hip_runtime.h>

// includes, project// helper functions for SDK examples
#include "tree.cuh"
#include <algorithm>
#include <fstream>
using namespace std;



bool findHost(int val)
{
	int* result = NULL;
	int host1[] = {2};
	hipError_t err = hipMalloc((void **)&result, 1 * sizeof(int));
	if (err != hipSuccess)
		cout<<"result "<<hipGetErrorString(err)<<endl;
	hipMemcpy(result, host1, 1*sizeof(int), hipMemcpyHostToDevice);
	cout<<host1[0]<<endl;
	while(host1[0] < 5)
	{
		searchBetter<<<(ORDER + BLOCKSIZE -1) / BLOCKSIZE, BLOCKSIZE>>>(val , result);
		err = hipDeviceSynchronize();
		hipMemcpy(host1, result, 1*sizeof(int),  hipMemcpyDeviceToHost);
		err = hipDeviceSynchronize();
		if(host1[0] > 0)
			cout<<host1[0]<<endl;
	}
	containsBetter<<<(ORDER + BLOCKSIZE -1) / BLOCKSIZE, BLOCKSIZE>>>(val, result);
	err = hipDeviceSynchronize();
	hipMemcpy(host1, result, 1*sizeof(int),  hipMemcpyDeviceToHost);
	err = hipDeviceSynchronize();
	if(host1[0] == 1)
	{
		cout<<"FOUND"<<endl;
		return true;
	}
	else
	{
		cout<<"NOT FOUND"<<endl;
		return false;
	}
}

void splitHost(node* node1)
{
	node* nn;
	createNewNode<<<(ORDER + BLOCKSIZE -1) / BLOCKSIZE, BLOCKSIZE>>>(nn);
	hipDeviceSynchronize();
	copyToNewNode<<<(ORDER + BLOCKSIZE -1) / BLOCKSIZE, BLOCKSIZE>>>(nn);
	hipDeviceSynchronize();
}

void insertHost(node* node)
{
	int* fullDevice = NULL;
	int fullHost[1];
	hipError_t err = hipMalloc((void **)&fullDevice, 1 * sizeof(int));
	hipDeviceSynchronize();
	copyNode<<<(ORDER + BLOCKSIZE -1) / BLOCKSIZE, BLOCKSIZE>>>(node, fullDevice);
	hipMemcpy(fullHost, fullDevice, 1*sizeof(int),  hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	addValue<<<(ORDER + BLOCKSIZE -1) / BLOCKSIZE, BLOCKSIZE>>>(node);
	hipDeviceSynchronize();
	if(fullHost[0])
		splitHost(node);
}

void splitHost(int val)
{
	node* newNode;
	createNewNode<<<(ORDER + BLOCKSIZE -1) / BLOCKSIZE, BLOCKSIZE>>>(newNode);
	hipDeviceSynchronize();
	copyToNewNode<<<(ORDER + BLOCKSIZE -1) / BLOCKSIZE, BLOCKSIZE>>>(newNode);
	hipDeviceSynchronize();
	insertHost(newNode);
}

void insertHost(int val)
{
	if(findHost(val))
		return;
	int* fullDevice = NULL;
	int fullHost[1];
	hipError_t err = hipMalloc((void **)&fullDevice, 1 * sizeof(int));
	hipDeviceSynchronize();
	copyNode<<<(ORDER + BLOCKSIZE -1) / BLOCKSIZE, BLOCKSIZE>>>(val, fullDevice);
	hipMemcpy(fullHost, fullDevice, 1*sizeof(int),  hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	addValue<<<(ORDER + BLOCKSIZE -1) / BLOCKSIZE, BLOCKSIZE>>>(val);
	hipDeviceSynchronize();
	if(fullHost[0])
		splitHost(val);
}

int main(int argc, char **argv)
{
	string filename="/home/piotr/Uczelnia/Cuda/BTree/testfile";
	int* initValues = (int*)malloc(INPUTSIZE*sizeof(int));
	node* root;
	ifstream file(filename.c_str());
	string line;
	int i =0;
	while(getline(file, line))
	{
		initValues[i] = atoi(line.c_str());
		i++;
	}
	sort(initValues, initValues+INPUTSIZE);
    int* input = NULL;
    int* result = NULL;
    int* results = NULL;
    int* values = NULL;
    int myArray[3] = { -1, 898386, 40156 };
    int host[1], hostResults[3];
    hipError_t err = hipMalloc((void **)&input, INPUTSIZE * sizeof(int));
    if (err != hipSuccess)
    	cout<<"input "<<hipGetErrorString(err)<<endl;
    err = hipMalloc((void **)&root, sizeof(node));
	if (err != hipSuccess)
		cout<<"root "<<hipGetErrorString(err)<<endl;
    err = hipMalloc((void **)&result, 1 * sizeof(int));
    if (err != hipSuccess)
    	cout<<"result "<<hipGetErrorString(err)<<endl;
    err = hipMalloc((void **)&results, 3 * sizeof(int));
	if (err != hipSuccess)
		cout<<"results "<<hipGetErrorString(err)<<endl;
	err = hipMalloc((void **)&values, 3 * sizeof(int));
	if (err != hipSuccess)
		cout<<"values "<<hipGetErrorString(err)<<endl;
    err = hipMemcpy(input, initValues, INPUTSIZE*sizeof(int), hipMemcpyHostToDevice);
	if (err != hipSuccess)
		cout<<"copy input "<<hipGetErrorString(err)<<endl;
	err = hipMemcpy(values, myArray, 3*sizeof(int), hipMemcpyHostToDevice);
	if (err != hipSuccess)
			cout<<"copy values "<<hipGetErrorString(err)<<endl;
	hipDeviceSetLimit(hipLimitMallocHeapSize, 512000000);
    buildLeaves<<<INPUTSIZE/BLOCKSIZE + 1, BLOCKSIZE>>>(root, input, result);
    err = hipDeviceSynchronize();
    if (err != hipSuccess)
    	cout<<"buildLeaves "<<hipGetErrorString(err)<<endl;
    fillLeaves<<<INPUTSIZE/BLOCKSIZE + 1, BLOCKSIZE>>>(root, input, result);
    err = hipDeviceSynchronize();
    if (err != hipSuccess)
    	cout<<"fillLeaves "<<hipGetErrorString(err)<<endl;
    int size = INPUTSIZE / ((ORDER/2)-1);
    int sw = 1;
    int level =0;
    while(size > ORDER)
    {
    	cout<<"Size of level "<<level<<": "<<size<<endl;
    	buildLevel<<<INPUTSIZE/BLOCKSIZE + 1, BLOCKSIZE>>>(root, input, result, size, sw);
    	err = hipDeviceSynchronize();
		if (err != hipSuccess)
			cout<<"buildlevel "<<hipGetErrorString(err)<<endl;
    	fillLevel<<<INPUTSIZE/BLOCKSIZE + 1, BLOCKSIZE>>>(root, input, result, size, sw);
    	err = hipDeviceSynchronize();
		if (err != hipSuccess)
			cout<<"filllevel "<<hipGetErrorString(err)<<endl;
    	size = size /(ORDER/2);
    	sw = 1 - sw;
    	level++;
    }
    cout<<"Size of level "<<level<<": "<<size<<endl;
    buildRoot<<<INPUTSIZE/BLOCKSIZE + 1, BLOCKSIZE>>>(root, input, result);
	err = hipDeviceSynchronize();
	if (err != hipSuccess)
		cout<<"build root "<<hipGetErrorString(err)<<endl;
	fillRoot<<<INPUTSIZE/BLOCKSIZE + 1, BLOCKSIZE>>>(root, input, result, size, sw);
	if (err != hipSuccess)
		cout<<"fill root "<<hipGetErrorString(err)<<endl;
	err = hipDeviceSynchronize();
    //test<<<(INPUTSIZE/BLOCKSIZE), BLOCKSIZE>>>(root, input, result);
	int v = -1;
	findHost(97861);
	insertHost(-1);
	findHost(-1);
	search<<<1, BLOCKSIZE>>>(v, result);
    err = hipDeviceSynchronize();
    if (err != hipSuccess)
       	cout<<"1st search call "<<hipGetErrorString(err)<<endl;
    hipMemcpy(host, result, 1*sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
        	cout<<"copying result "<<hipGetErrorString(err)<<endl;

    int f = 0;
    for(int i=0; i<INPUTSIZE; i++)
    {
    	if(initValues[i] == v)
    		f = 1;
    }
    cout<<host[0]<<" "<<f<<endl;
    search<<<3, BLOCKSIZE>>>(values, results, 3);
    err = hipDeviceSynchronize();
    if (err != hipSuccess)
    		cout<<"multiple search call "<<hipGetErrorString(err)<<endl;
    hipMemcpy(hostResults, results, 3*sizeof(int), hipMemcpyDeviceToHost);
    for(int i=0; i<3; i++)
    	cout<<hostResults[i]<<endl;
    insertVal<<<1, BLOCKSIZE>>>(v);
    err = hipDeviceSynchronize();
    if (err != hipSuccess)
    	cout<<"insert call "<<hipGetErrorString(err)<<endl;
    search<<<1, BLOCKSIZE>>>(v, result);
    err = hipDeviceSynchronize();
    if (err != hipSuccess)
		cout<<"2nd search call "<<hipGetErrorString(err)<<endl;
    hipMemcpy(host, result, 1*sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    	cout<<"copying result "<<hipGetErrorString(err)<<endl;
    cout<<host[0]<<endl;
    cout<<"end"<<endl;
    hipFree(input);
    hipFree(result);
}
